
#include <hip/hip_runtime.h>
#include<cstdio>
#include<cstdlib>
#define SIZE 128

__global__ void simple_test(int *arr){
    arr[threadIdx.x] = SIZE - threadIdx.x;
}
int main(){
    int *arr_h,*arr_d;
    arr_h = (int*)malloc(SIZE*sizeof(int));
    hipMalloc(&arr_d,SIZE*sizeof(int));

    simple_test<<<1,SIZE>>>(arr_d);

    hipMemcpy(arr_h,arr_d,SIZE*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<SIZE;i++){
        if(arr_h[i]!=SIZE-i){
            printf("test2:check fail\n");
            goto Free;
        }
    }
    printf("test2:check pass\n");
Free:
    hipFree(arr_d);
    free(arr_h);
}